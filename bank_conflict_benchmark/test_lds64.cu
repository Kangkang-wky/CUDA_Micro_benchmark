
#include <hip/hip_runtime.h>
#include <cstdint>

// lds64 指令测试

__global__ void smem_1(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  if (tid < 16) {
    reinterpret_cast<uint2 *>(a)[tid] =
        reinterpret_cast<const uint2 *>(smem)[tid];
  }
}

__global__ void smem_2(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  if (tid < 15 || tid == 16) {
    reinterpret_cast<uint2 *>(a)[tid] =
        reinterpret_cast<const uint2 *>(smem)[tid == 16 ? 15 : tid];
  }
}

__global__ void smem_3(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  reinterpret_cast<uint2 *>(a)[tid] =
      reinterpret_cast<const uint2 *>(smem)[tid / 2];
}

__global__ void smem_4(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  uint32_t addr;
  if (tid < 16) {
    addr = tid / 2;
  } else {
    addr = (tid / 4) * 4 + (tid % 4) % 2;
  }
  reinterpret_cast<uint2 *>(a)[tid] =
      reinterpret_cast<const uint2 *>(smem)[addr];
}

__global__ void smem_5(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  reinterpret_cast<uint2 *>(a)[tid] =
      reinterpret_cast<const uint2 *>(smem)[tid % 16];
}


int main() {
  uint32_t *d_a;
  hipMalloc(&d_a, sizeof(uint32_t) * 128);
  // micro benchmark
  smem_1<<<1, 32>>>(d_a);
  smem_2<<<1, 32>>>(d_a);
  smem_3<<<1, 32>>>(d_a);
  smem_4<<<1, 32>>>(d_a);
  smem_5<<<1, 32>>>(d_a);
  hipFree(d_a);
  hipDeviceSynchronize();
  return 0;
}